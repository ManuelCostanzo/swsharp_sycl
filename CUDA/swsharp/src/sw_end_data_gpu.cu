#include "hip/hip_runtime.h"
/*
This code represents a SYCL-compatible, DPC++-based version of SW#.
Copyright (C) 2022 Manuel Costanzo, contributor Enzo Rucci.

swsharp - CUDA parallelized Smith Waterman with applying Hirschberg's and
Ukkonen's algorithm and dynamic cell pruning.
Copyright (C) 2013 Matija Korpar, contributor Mile Šikić

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact SW# author by mkorpar@gmail.com.

Contact SW#-SYCL authors by mcostanzo@lidi.info.unlp.edu.ar, erucci@lidi.info.unlp.edu.ar
*/

#ifdef __HIPCC__

#include <stdlib.h>
#include <stdio.h>

#include "chain.h"
#include "constants.h"
#include "cuda_utils.h"
#include "error.h"
#include "scorer.h"
#include "thread.h"
#include "utils.h"

#include "gpu_module.h"

#define MAX_THREADS 1024

#define THREADS_SM1 64
#define BLOCKS_SM1 240

#define THREADS_SM2 128
#define BLOCKS_SM2 480

#define INT4_ZERO make_int4(0, 0, 0, 0)

typedef struct Atom
{
    int mch;
    int2 up;
    int4 lScr;
    int4 lAff;
    int4 rScr;
    int4 rAff;
} Atom;

typedef struct VBus
{
    int *mch;
    int4 *scr;
    int4 *aff;
} VBus;

typedef struct Context
{
    int **scores;
    int **affines;
    int *queryEnd;
    int *targetEnd;
    int *outScore;
    Chain *query;
    Chain *target;
    Scorer *scorer;
    int score;
    int card;
} Context;

static __constant__ int gapOpen_;
static __constant__ int gapExtend_;

static __constant__ int rows_;
static __constant__ int cols_;

static __constant__ int cellWidth_;

static __constant__ int pruneLow_;
static __constant__ int pruneHigh_;

static __constant__ int scorerLen_;
static __constant__ int subLen_;

static __constant__ int match_;
static __constant__ int mismatch_;

texture<char4> rowTexture;
texture<char> colTexture;
texture<int2> hBusTexture;
texture<int> subTexture;

//******************************************************************************
// PUBLIC

extern void swEndDataGpu(int *queryEnd, int *targetEnd, int *outScore,
                         int **scores, int **affines, Chain *query, Chain *target, Scorer *scorer,
                         int score, int card, Thread *thread);

//******************************************************************************

//******************************************************************************
// PRIVATE

// With visual c++ compiler and prototypes declared cuda global memory variables
// do not work. No questions asked.
#ifndef _WIN32

template <class Sub>
__device__ static void solveShortDelegated(int d, VBus vBus, int2 *hBus,
                                           int3 *results, Sub sub);

template <class Sub>
__device__ static void solveShortNormal(int d, VBus vBus, int2 *hBus,
                                        int3 *results, Sub sub);

template <class Sub>
__global__ static void solveShort(int d, VBus vBus, int2 *hBus, int3 *results,
                                  Sub sub);

template <class Sub>
__global__ static void solveLong(int d, VBus vBus, int2 *hBus, int *bBus,
                                 int3 *results, Sub sub);

#endif

static void *kernel(void *params);

//******************************************************************************

//******************************************************************************
// PUBLIC

extern void swEndDataGpu(int *queryEnd, int *targetEnd, int *outScore,
                         int **scores, int **affines, Chain *query, Chain *target, Scorer *scorer,
                         int score, int card, Thread *thread)
{

    Context *param = (Context *)malloc(sizeof(Context));

    param->scores = scores;
    param->affines = affines;
    param->queryEnd = queryEnd;
    param->targetEnd = targetEnd;
    param->outScore = outScore;
    param->query = query;
    param->target = target;
    param->scorer = scorer;
    param->score = score;
    param->card = card;

    if (thread == NULL)
    {
        kernel(param);
    }
    else
    {
        threadCreate(thread, kernel, (void *)param);
    }
}

//******************************************************************************

//******************************************************************************
// PRIVATE

//------------------------------------------------------------------------------
// FUNCTORS

class SubScalar
{
public:
    __device__ int operator()(char a, char b)
    {
        return a == b ? match_ : mismatch_;
    }
};

class SubScalarRev
{
public:
    __device__ int operator()(char a, char b)
    {
        return (a == b ? match_ : mismatch_) * (a < scorerLen_ && b < scorerLen_);
    }
};

class SubVector
{
public:
    __device__ int operator()(char a, char b)
    {
        return tex1Dfetch(subTexture, (a * subLen_) + b);
    }
};

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// GPU KERNELS

template <class Sub>
__device__ static void solveShortDelegated(int d, VBus vBus, int2 *hBus,
                                           int3 *results, Sub sub)
{

    __shared__ int hBusScrShr[MAX_THREADS];
    __shared__ int hBusAffShr[MAX_THREADS];

    if (pruneLow_ >= 0 && pruneHigh_ < gridDim.x)
    {
        return;
    }

    int row = (d + blockIdx.x - gridDim.x + 1) * (blockDim.x * 4) + threadIdx.x * 4;
    int col = cellWidth_ * (gridDim.x - blockIdx.x - 1) - threadIdx.x;

    if (row < 0)
        return;

    row -= (col < 0) * (gridDim.x * blockDim.x * 4);
    col += (col < 0) * cols_;

    Atom atom;

    if (0 <= row && row < rows_ && col > 0)
    {
        atom.mch = vBus.mch[(row >> 2) % (gridDim.x * blockDim.x)];
        VEC4_ASSIGN(atom.lScr, vBus.scr[(row >> 2) % (gridDim.x * blockDim.x)]);
        VEC4_ASSIGN(atom.lAff, vBus.aff[(row >> 2) % (gridDim.x * blockDim.x)]);
    }
    else
    {
        atom.mch = 0;
        VEC4_ASSIGN(atom.lScr, INT4_ZERO);
        VEC4_ASSIGN(atom.lAff, INT4_ZERO);
    }

    hBusScrShr[threadIdx.x] = tex1Dfetch(hBusTexture, col).x;
    hBusAffShr[threadIdx.x] = tex1Dfetch(hBusTexture, col).y;

    char4 rowCodes = tex1Dfetch(rowTexture, row >> 2);
    int3 res = {0, 0, 0};

    int del;

    for (int i = 0; i < blockDim.x; ++i)
    {

        if (0 <= row && row < rows_)
        {

            char columnCode = tex1Dfetch(colTexture, col);

            if (threadIdx.x == 0)
            {
                atom.up = tex1Dfetch(hBusTexture, col);
            }
            else
            {
                atom.up.x = hBusScrShr[threadIdx.x];
                atom.up.y = hBusAffShr[threadIdx.x];
            }

            del = max(atom.up.x - gapOpen_, atom.up.y - gapExtend_);
            int ins = max(atom.lScr.x - gapOpen_, atom.lAff.x - gapExtend_);
            int mch = atom.mch + sub(columnCode, rowCodes.x);

            atom.rScr.x = MAX4(0, mch, del, ins);
            atom.rAff.x = ins;

            del = max(atom.rScr.x - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.y - gapOpen_, atom.lAff.y - gapExtend_);
            mch = atom.lScr.x + sub(columnCode, rowCodes.y);

            atom.rScr.y = MAX4(0, mch, del, ins);
            atom.rAff.y = ins;

            del = max(atom.rScr.y - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.z - gapOpen_, atom.lAff.z - gapExtend_);
            mch = atom.lScr.y + sub(columnCode, rowCodes.z);

            atom.rScr.z = MAX4(0, mch, del, ins);
            atom.rAff.z = ins;

            del = max(atom.rScr.z - gapOpen_, del - gapExtend_);
            ins = max(atom.lScr.w - gapOpen_, atom.lAff.w - gapExtend_);
            mch = atom.lScr.z + sub(columnCode, rowCodes.w);

            atom.rScr.w = MAX4(0, mch, del, ins);
            atom.rAff.w = ins;

            if (atom.rScr.x > res.x)
            {
                res.x = atom.rScr.x;
                res.y = row;
                res.z = col;
            }
            if (atom.rScr.y > res.x)
            {
                res.x = atom.rScr.y;
                res.y = row + 1;
                res.z = col;
            }
            if (atom.rScr.z > res.x)
            {
                res.x = atom.rScr.z;
                res.y = row + 2;
                res.z = col;
            }
            if (atom.rScr.w > res.x)
            {
                res.x = atom.rScr.w;
                res.y = row + 3;
                res.z = col;
            }

            atom.mch = atom.up.x;
            VEC4_ASSIGN(atom.lScr, atom.rScr);
            VEC4_ASSIGN(atom.lAff, atom.rAff);
        }

        __syncthreads();

        if (0 <= row && row < rows_)
        {

            if (threadIdx.x == blockDim.x - 1 || i == blockDim.x - 1)
            {
                VEC2_ASSIGN(hBus[col], make_int2(atom.rScr.w, del));
            }
            else
            {
                hBusScrShr[threadIdx.x + 1] = atom.rScr.w;
                hBusAffShr[threadIdx.x + 1] = del;
            }
        }

        ++col;

        if (col == cols_)
        {

            col = 0;
            row = row + gridDim.x * blockDim.x * 4;

            atom.mch = 0;
            VEC4_ASSIGN(atom.lScr, INT4_ZERO);
            atom.lAff = atom.lScr;

            rowCodes = tex1Dfetch(rowTexture, row >> 2);
        }

        __syncthreads();
    }

    if (res.x > results[blockIdx.x * blockDim.x + threadIdx.x].x)
    {
        VEC3_ASSIGN(results[blockIdx.x * blockDim.x + threadIdx.x], res);
    }

    if (row < 0 || row >= rows_)
        return;

    vBus.mch[(row >> 2) % (gridDim.x * blockDim.x)] = atom.up.x;
    VEC4_ASSIGN(vBus.scr[(row >> 2) % (gridDim.x * blockDim.x)], atom.lScr);
    VEC4_ASSIGN(vBus.aff[(row >> 2) % (gridDim.x * blockDim.x)], atom.lAff);
}

template <class Sub>
__device__ static void solveShortNormal(int d, VBus vBus, int2 *hBus,
                                        int3 *results, Sub sub)
{

    __shared__ int hBusScrShr[MAX_THREADS];
    __shared__ int hBusAffShr[MAX_THREADS];

    if ((int)blockIdx.x <= pruneLow_ || blockIdx.x >= pruneHigh_)
    {
        return;
    }

    int row = (d + blockIdx.x - gridDim.x + 1) * (blockDim.x * 4) + threadIdx.x * 4;
    int col = cellWidth_ * (gridDim.x - blockIdx.x - 1) - threadIdx.x;

    if (row < 0 || row >= rows_)
        return;

    Atom atom;
    atom.mch = vBus.mch[(row >> 2) % (gridDim.x * blockDim.x)];
    VEC4_ASSIGN(atom.lScr, vBus.scr[(row >> 2) % (gridDim.x * blockDim.x)]);
    VEC4_ASSIGN(atom.lAff, vBus.aff[(row >> 2) % (gridDim.x * blockDim.x)]);

    hBusScrShr[threadIdx.x] = tex1Dfetch(hBusTexture, col).x;
    hBusAffShr[threadIdx.x] = tex1Dfetch(hBusTexture, col).y;

    const char4 rowCodes = tex1Dfetch(rowTexture, row >> 2);
    int3 res = {0, 0, 0};

    int del;

    for (int i = 0; i < blockDim.x; ++i, ++col)
    {

        char columnCode = tex1Dfetch(colTexture, col);

        if (threadIdx.x == 0)
        {
            atom.up = tex1Dfetch(hBusTexture, col);
        }
        else
        {
            atom.up = make_int2(hBusScrShr[threadIdx.x], hBusAffShr[threadIdx.x]);
        }

        del = max(atom.up.x - gapOpen_, atom.up.y - gapExtend_);
        int ins = max(atom.lScr.x - gapOpen_, atom.lAff.x - gapExtend_);
        int mch = atom.mch + sub(columnCode, rowCodes.x);

        atom.rScr.x = MAX4(0, mch, del, ins);
        atom.rAff.x = ins;

        del = max(atom.rScr.x - gapOpen_, del - gapExtend_);
        ins = max(atom.lScr.y - gapOpen_, atom.lAff.y - gapExtend_);
        mch = atom.lScr.x + sub(columnCode, rowCodes.y);

        atom.rScr.y = MAX4(0, mch, del, ins);
        atom.rAff.y = ins;

        del = max(atom.rScr.y - gapOpen_, del - gapExtend_);
        ins = max(atom.lScr.z - gapOpen_, atom.lAff.z - gapExtend_);
        mch = atom.lScr.y + sub(columnCode, rowCodes.z);

        atom.rScr.z = MAX4(0, mch, del, ins);
        atom.rAff.z = ins;

        del = max(atom.rScr.z - gapOpen_, del - gapExtend_);
        ins = max(atom.lScr.w - gapOpen_, atom.lAff.w - gapExtend_);
        mch = atom.lScr.z + sub(columnCode, rowCodes.w);

        atom.rScr.w = MAX4(0, mch, del, ins);
        atom.rAff.w = ins;

        if (atom.rScr.x > res.x)
        {
            res.x = atom.rScr.x;
            res.y = row;
            res.z = col;
        }
        if (atom.rScr.y > res.x)
        {
            res.x = atom.rScr.y;
            res.y = row + 1;
            res.z = col;
        }
        if (atom.rScr.z > res.x)
        {
            res.x = atom.rScr.z;
            res.y = row + 2;
            res.z = col;
        }
        if (atom.rScr.w > res.x)
        {
            res.x = atom.rScr.w;
            res.y = row + 3;
            res.z = col;
        }

        atom.mch = atom.up.x;
        VEC4_ASSIGN(atom.lScr, atom.rScr);
        VEC4_ASSIGN(atom.lAff, atom.rAff);

        __syncthreads();

        if (threadIdx.x == blockDim.x - 1)
        {
            VEC2_ASSIGN(hBus[col], make_int2(atom.rScr.w, del));
        }
        else
        {
            hBusScrShr[threadIdx.x + 1] = atom.rScr.w;
            hBusAffShr[threadIdx.x + 1] = del;
        }

        __syncthreads();
    }

    const int vBusIdx = (row >> 2) % (gridDim.x * blockDim.x);
    vBus.mch[vBusIdx] = atom.up.x;
    VEC4_ASSIGN(vBus.scr[vBusIdx], atom.lScr);
    VEC4_ASSIGN(vBus.aff[vBusIdx], atom.lAff);

    VEC2_ASSIGN(hBus[col - 1], make_int2(atom.rScr.w, del));

    if (res.x > results[blockIdx.x * blockDim.x + threadIdx.x].x)
    {
        VEC3_ASSIGN(results[blockIdx.x * blockDim.x + threadIdx.x], res);
    }
}

template <class Sub>
__global__ static void solveShort(int d, VBus vBus, int2 *hBus, int3 *results, Sub sub)
{

    if (blockIdx.x == (gridDim.x - 1))
    {
        solveShortDelegated(d, vBus, hBus, results, sub);
    }
    else
    {
        solveShortNormal(d, vBus, hBus, results, sub);
    }
}

template <class Sub>
__global__ static void solveLong(int d, VBus vBus, int2 *hBus, int *bBus,
                                 int3 *results, Sub sub)
{

    __shared__ int hBusScrShr[MAX_THREADS];
    __shared__ int hBusAffShr[MAX_THREADS];

    hBusScrShr[threadIdx.x] = 0;

    if ((int)blockIdx.x <= pruneLow_ || blockIdx.x > pruneHigh_)
    {
        return;
    }

    int row = (d + blockIdx.x - gridDim.x + 1) * (blockDim.x * 4) + threadIdx.x * 4;
    int col = cellWidth_ * (gridDim.x - blockIdx.x - 1) - threadIdx.x + blockDim.x;

    if (row < 0 || row >= rows_)
        return;

    if (blockIdx.x == pruneHigh_)
    {

        // clear only the last steepness
        vBus.mch[(row >> 2) % (gridDim.x * blockDim.x)] = 0;
        vBus.scr[(row >> 2) % (gridDim.x * blockDim.x)] = INT4_ZERO;
        vBus.aff[(row >> 2) % (gridDim.x * blockDim.x)] = INT4_ZERO;

        VEC2_ASSIGN(hBus[col + cellWidth_ - blockDim.x - 1], make_int2(0, 0));

        return;
    }

    Atom atom;
    atom.mch = vBus.mch[(row >> 2) % (gridDim.x * blockDim.x)];
    VEC4_ASSIGN(atom.lScr, vBus.scr[(row >> 2) % (gridDim.x * blockDim.x)]);
    VEC4_ASSIGN(atom.lAff, vBus.aff[(row >> 2) % (gridDim.x * blockDim.x)]);

    hBusScrShr[threadIdx.x] = tex1Dfetch(hBusTexture, col).x;
    hBusAffShr[threadIdx.x] = tex1Dfetch(hBusTexture, col).y;

    const char4 rowCodes = tex1Dfetch(rowTexture, row >> 2);
    int3 res = {0, 0, 0};

    int del;

    for (int i = 0; i < cellWidth_ - blockDim.x; ++i, ++col)
    {

        char columnCode = tex1Dfetch(colTexture, col);

        if (threadIdx.x == 0)
        {
            atom.up = tex1Dfetch(hBusTexture, col);
        }
        else
        {
            atom.up = make_int2(hBusScrShr[threadIdx.x], hBusAffShr[threadIdx.x]);
        }

        del = max(atom.up.x - gapOpen_, atom.up.y - gapExtend_);
        int ins = max(atom.lScr.x - gapOpen_, atom.lAff.x - gapExtend_);
        int mch = atom.mch + sub(columnCode, rowCodes.x);

        atom.rScr.x = MAX4(0, mch, del, ins);
        atom.rAff.x = ins;

        del = max(atom.rScr.x - gapOpen_, del - gapExtend_);
        ins = max(atom.lScr.y - gapOpen_, atom.lAff.y - gapExtend_);
        mch = atom.lScr.x + sub(columnCode, rowCodes.y);

        atom.rScr.y = MAX4(0, mch, del, ins);
        atom.rAff.y = ins;

        del = max(atom.rScr.y - gapOpen_, del - gapExtend_);
        ins = max(atom.lScr.z - gapOpen_, atom.lAff.z - gapExtend_);
        mch = atom.lScr.y + sub(columnCode, rowCodes.z);

        atom.rScr.z = MAX4(0, mch, del, ins);
        atom.rAff.z = ins;

        del = max(atom.rScr.z - gapOpen_, del - gapExtend_);
        ins = max(atom.lScr.w - gapOpen_, atom.lAff.w - gapExtend_);
        mch = atom.lScr.z + sub(columnCode, rowCodes.w);

        atom.rScr.w = MAX4(0, mch, del, ins);
        atom.rAff.w = ins;

        if (atom.rScr.x > res.x)
        {
            res.x = atom.rScr.x;
            res.y = row;
            res.z = col;
        }
        if (atom.rScr.y > res.x)
        {
            res.x = atom.rScr.y;
            res.y = row + 1;
            res.z = col;
        }
        if (atom.rScr.z > res.x)
        {
            res.x = atom.rScr.z;
            res.y = row + 2;
            res.z = col;
        }
        if (atom.rScr.w > res.x)
        {
            res.x = atom.rScr.w;
            res.y = row + 3;
            res.z = col;
        }

        atom.mch = atom.up.x;
        VEC4_ASSIGN(atom.lScr, atom.rScr);
        VEC4_ASSIGN(atom.lAff, atom.rAff);

        __syncthreads();

        if (threadIdx.x == blockDim.x - 1)
        {
            VEC2_ASSIGN(hBus[col], make_int2(atom.rScr.w, del));
        }
        else
        {
            hBusScrShr[threadIdx.x + 1] = atom.rScr.w;
            hBusAffShr[threadIdx.x + 1] = del;
        }

        __syncthreads();
    }

    const int vBusIdx = (row >> 2) % (gridDim.x * blockDim.x);
    vBus.mch[vBusIdx] = atom.up.x;
    VEC4_ASSIGN(vBus.scr[vBusIdx], atom.lScr);
    VEC4_ASSIGN(vBus.aff[vBusIdx], atom.lAff);

    VEC2_ASSIGN(hBus[col - 1], make_int2(atom.rScr.w, del));

    if (res.x > results[blockIdx.x * blockDim.x + threadIdx.x].x)
    {
        VEC3_ASSIGN(results[blockIdx.x * blockDim.x + threadIdx.x], res);
    }

    // reuse
    hBusScrShr[threadIdx.x] = res.x;
    __syncthreads();

    int score = 0;
    int idx = 0;

    for (int i = 0; i < blockDim.x; ++i)
    {

        int shr = hBusScrShr[i];

        if (shr > score)
        {
            score = shr;
            idx = i;
        }
    }

    if (threadIdx.x == idx)
        bBus[blockIdx.x] = score;
}

//------------------------------------------------------------------------------

//------------------------------------------------------------------------------
// CPU KERNELS

static void *kernel(void *params)
{

    Context *context = (Context *)params;

    int **scores = context->scores;
    int **affines = context->affines;
    int *queryEnd = context->queryEnd;
    int *targetEnd = context->targetEnd;
    int *outScore = context->outScore;
    Chain *query = context->query;
    Chain *target = context->target;
    Scorer *scorer = context->scorer;
    int score = context->score;
    int card = context->card;

    // if negative matrix, no need for SW, score will not be found
    if (scorerGetMaxScore(scorer) <= 0)
    {
        *outScore = NO_SCORE;
        *queryEnd = 0;
        *targetEnd = 0;
        if (scores != NULL)
            *scores = NULL;
        if (affines != NULL)
            *affines = NULL;
        free(params);
        return NULL;
    }

    int currentCard;
    CUDA_SAFE_CALL(hipGetDevice(&currentCard));
    if (currentCard != card)
    {
        // CUDA_SAFE_CALL(hipDeviceReset());
        CUDA_SAFE_CALL(hipSetDevice(card));
    }

    int rows = chainGetLength(query);
    int cols = chainGetLength(target);
    int gapOpen = scorerGetGapOpen(scorer);
    int gapExtend = scorerGetGapExtend(scorer);
    int scorerLen = scorerGetMaxCode(scorer);
    int subLen = scorerLen + 1;
    int scalar = scorerIsScalar(scorer);

    TIMER_START("Sw end data %d %d", rows, cols);

    hipDeviceProp_t properties;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&properties, card));

    int threads;
    int blocks;

    maxWorkGroups(card, BLOCKS_SM2, THREADS_SM2, cols, &blocks, &threads);

    int cellHeight = 4 * threads;
    int rowsGpu = rows + (cellHeight - rows % cellHeight) % cellHeight;

    int colsGpu = cols + (blocks - cols % blocks) % blocks;
    int cellWidth = colsGpu / blocks;

    int diagonals = blocks + (rowsGpu / cellHeight);

    int pruneLow = -1;
    int pruneHigh = blocks;
    int pruneFactor = scorerGetMaxScore(scorer);

    int memoryUsedGpu = 0;
    int memoryUsedCpu = 0;

    /*
    LOG("Rows cpu: %d, gpu: %d", rows, rowsGpu);
    LOG("Columns cpu: %d, gpu: %d", cols, colsGpu);
    LOG("Cell h: %d, w: %d", cellHeight, cellWidth);
    LOG("Diagonals: %d", diagonals);
    */

    //**************************************************************************
    // PADD CHAINS
    char *rowCpu = (char *)malloc(rowsGpu * sizeof(char));
    memset(rowCpu, scorerLen, (rowsGpu - rows) * sizeof(char));
    chainCopyCodes(query, rowCpu + (rowsGpu - rows));
    memoryUsedCpu += rowsGpu * sizeof(char);

    char *colCpu = (char *)malloc(colsGpu * sizeof(char));
    memset(colCpu + cols, scorerLen + scalar, (colsGpu - cols) * sizeof(char));
    chainCopyCodes(target, colCpu);
    memoryUsedCpu += colsGpu * sizeof(char);
    //**************************************************************************

    //**************************************************************************
    // INIT GPU
    size_t rowSize = rowsGpu * sizeof(char);
    char4 *rowGpu;
    CUDA_SAFE_CALL(hipMalloc(&rowGpu, rowSize));
    CUDA_SAFE_CALL(hipMemcpy(rowGpu, rowCpu, rowSize, TO_GPU));
    CUDA_SAFE_CALL(hipBindTexture(NULL, rowTexture, rowGpu, rowSize));
    memoryUsedGpu += rowSize;

    size_t colSize = colsGpu * sizeof(char);
    char *colGpu;
    CUDA_SAFE_CALL(hipMalloc(&colGpu, colSize));
    CUDA_SAFE_CALL(hipMemcpy(colGpu, colCpu, colSize, TO_GPU));
    CUDA_SAFE_CALL(hipBindTexture(NULL, colTexture, colGpu, colSize));
    memoryUsedGpu += colSize;

    size_t hBusSize = colsGpu * sizeof(int2);
    int2 *hBusCpu;
    int2 *hBusGpu;
    CUDA_SAFE_CALL(hipHostMalloc(&hBusCpu, hBusSize));
    CUDA_SAFE_CALL(hipMalloc(&hBusGpu, hBusSize));
    CUDA_SAFE_CALL(hipMemset(hBusGpu, 0, hBusSize));
    CUDA_SAFE_CALL(hipBindTexture(NULL, hBusTexture, hBusGpu, hBusSize));
    memoryUsedCpu += hBusSize;
    memoryUsedGpu += hBusSize;

    VBus vBusGpu;
    CUDA_SAFE_CALL(hipMalloc(&vBusGpu.mch, blocks * threads * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc(&vBusGpu.scr, blocks * threads * sizeof(int4)));
    CUDA_SAFE_CALL(hipMalloc(&vBusGpu.aff, blocks * threads * sizeof(int4)));
    memoryUsedGpu += blocks * threads * sizeof(int);
    memoryUsedGpu += blocks * threads * sizeof(int4);
    memoryUsedGpu += blocks * threads * sizeof(int4);

    size_t resultsSize = blocks * threads * sizeof(int3);
    int3 *resultsCpu = (int3 *)malloc(resultsSize);
    int3 *resultsGpu;
    CUDA_SAFE_CALL(hipMalloc(&resultsGpu, resultsSize));
    CUDA_SAFE_CALL(hipMemset(resultsGpu, 0, resultsSize));
    memoryUsedCpu += resultsSize;
    memoryUsedGpu += resultsSize;

    size_t bSize = blocks * sizeof(int);
    int *bCpu;
    int *bGpu;
    CUDA_SAFE_CALL(hipHostMalloc(&bCpu, bSize));
    CUDA_SAFE_CALL(hipMalloc(&bGpu, bSize));
    CUDA_SAFE_CALL(hipMemset(bGpu, 0, bSize));
    memoryUsedCpu += bSize;
    memoryUsedGpu += bSize;

    size_t subSize = subLen * subLen * sizeof(int);
    int *subCpu = (int *)malloc(subSize);
    int *subGpu;
    for (int i = 0; i < subLen; ++i)
    {
        for (int j = 0; j < subLen; ++j)
        {
            if (i < scorerLen && j < scorerLen)
            {
                subCpu[i * subLen + j] = scorerScore(scorer, i, j);
            }
            else
            {
                subCpu[i * subLen + j] = 0;
            }
        }
    }
    CUDA_SAFE_CALL(hipMalloc(&subGpu, subSize));
    CUDA_SAFE_CALL(hipMemcpy(subGpu, subCpu, subSize, TO_GPU));
    CUDA_SAFE_CALL(hipBindTexture(NULL, subTexture, subGpu, subSize));
    memoryUsedCpu += subSize;
    memoryUsedGpu += subSize;

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(match_), &(subCpu[0]), sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(mismatch_), &(subCpu[1]), sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gapOpen_), &gapOpen, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(gapExtend_), &gapExtend, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(scorerLen_), &scorerLen, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(subLen_), &subLen, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(rows_), &rowsGpu, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cols_), &colsGpu, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cellWidth_), &cellWidth, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pruneLow_), &pruneLow, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pruneHigh_), &pruneHigh, sizeof(int)));

    // LOG("Memory used CPU: %fMB", memoryUsedCpu / 1024. / 1024.);
    LOG("Memory used GPU: %fMB", memoryUsedGpu / 1024. / 1024.);

    //**************************************************************************

    //**************************************************************************
    // KERNEL RUN

    int best = MAX(0, score);
    int pruning = 1;
    int pruned = 0;
    int pruneHighOld = pruneHigh;
    int halfPruning = scores != NULL || affines != NULL;

    // TIMER_START("Kernel");

    for (int diagonal = 0; diagonal < diagonals; ++diagonal)
    {

        if (scalar)
        {
            if (subCpu[0] >= subCpu[1])
            {
                solveShort<<<blocks, threads>>>(diagonal, vBusGpu, hBusGpu, resultsGpu, SubScalar());
                solveLong<<<blocks, threads>>>(diagonal, vBusGpu, hBusGpu, bGpu, resultsGpu, SubScalar());
            }
            else
            {
                // cannot use mismatch negative trick
                solveShort<<<blocks, threads>>>(diagonal, vBusGpu, hBusGpu, resultsGpu, SubScalarRev());
                solveLong<<<blocks, threads>>>(diagonal, vBusGpu, hBusGpu, bGpu, resultsGpu, SubScalarRev());
            }
        }
        else
        {
            solveShort<<<blocks, threads>>>(diagonal, vBusGpu, hBusGpu, resultsGpu, SubVector());
            solveLong<<<blocks, threads>>>(diagonal, vBusGpu, hBusGpu, bGpu, resultsGpu, SubVector());
        }

        if (pruning)
        {

            size_t bSize = pruneHigh * sizeof(int);
            CUDA_SAFE_CALL(hipMemcpy(bCpu, bGpu, bSize, FROM_GPU));

            if (score == NO_SCORE)
            {
                for (int i = 0; i < pruneHigh; ++i)
                {
                    best = max(best, bCpu[i]);
                }
            }

            // delta j pruning
            pruneLow = -1;
            for (int i = 0; i < blocks; ++i)
            {
                int row = (diagonal + 1 + i - blocks + 1) * (threads * 4);
                int col = cellWidth * (blocks - i - 1) - threads;
                if (row >= rowsGpu)
                    break;
                if (rowsGpu * (halfPruning ? 2 : 1) - row < cols - col)
                    break;
                int d = cols - col;
                int scr = i == blocks - 1 ? bCpu[i] : max(bCpu[i], bCpu[i + 1]);
                if ((scr + d * pruneFactor) < best)
                    pruneLow = i;
                else
                    break;
            }

            // delta i pruning
            if (!halfPruning)
            {
                pruneHighOld = pruneHigh;
                for (int i = pruneHighOld - 1; i >= 0; --i)
                {
                    int row = (diagonal + 1 + i - blocks + 1) * (threads * 4);
                    int col = cellWidth * (blocks - i - 1) - threads;
                    if (row < rowsGpu / 2)
                        break;
                    if (row >= rowsGpu)
                        continue;
                    if (rowsGpu - row > cols - col)
                        break;
                    int d = rowsGpu - row;
                    int scr1 = d * pruneFactor + (i == blocks - 1 ? 0 : bCpu[i + 1]);
                    int scr2 = (d + threads * 2) * pruneFactor + bCpu[i];
                    if (scr1 < best && scr2 < best)
                        pruneHigh = i;
                    else
                        break;
                }
            }

            pruned += blocks - (pruneHigh - pruneLow - 1);

            if (pruneLow >= pruneHigh)
            {
                break;
            }

            CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pruneLow_), &pruneLow, sizeof(int)));
            CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pruneHigh_), &pruneHigh, sizeof(int)));

            if (pruneLow >= 0)
            {
                int offset = (blocks - pruneLow - 1) * cellWidth - threads;
                size_t size = (colsGpu - offset) * sizeof(int2);
                CUDA_SAFE_CALL(hipMemset(hBusGpu + offset, 0, size));
            }
        }
    }

    // TIMER_STOP;

    LOG("Pruned percentage %.2f%%", 100.0 * pruned / (diagonals * blocks));

    //**************************************************************************

    //**************************************************************************
    // SAVE RESULTS

    // save only if needed
    if (scores != NULL && affines != NULL)
    {

        CUDA_SAFE_CALL(hipMemcpy(hBusCpu, hBusGpu, hBusSize, FROM_GPU));

        *scores = (int *)malloc(cols * sizeof(int));
        *affines = (int *)malloc(cols * sizeof(int));

        for (int i = 0; i < cols; ++i)
        {
            (*scores)[i] = hBusCpu[i].x;
            (*affines)[i] = hBusCpu[i].y;
        }
    }

    CUDA_SAFE_CALL(hipMemcpy(resultsCpu, resultsGpu, resultsSize, FROM_GPU));

    int3 res = resultsCpu[0];
    for (int i = 1; i < blocks * threads; ++i)
    {
        if (resultsCpu[i].x > res.x)
        {
            res = resultsCpu[i];
        }
    }

    res.y -= (rowsGpu - rows); // restore padding

    // check if the result updated in the padded part
    if (res.y >= rows)
    {
        res.z += rows - res.y - 1;
        res.y += rows - res.y - 1;
    }

    if (res.z >= cols)
    {
        res.y += cols - res.z - 1;
        res.z += cols - res.z - 1;
    }

    *outScore = res.x;
    *queryEnd = res.y;
    *targetEnd = res.z;

    LOG("Score: %d, (%d, %d)", *outScore, *queryEnd, *targetEnd);

    //**************************************************************************

    //**************************************************************************
    // CLEAN MEMORY

    free(subCpu);
    free(rowCpu);
    free(colCpu);
    free(resultsCpu);

    CUDA_SAFE_CALL(hipHostFree(bCpu));
    CUDA_SAFE_CALL(hipHostFree(hBusCpu));

    CUDA_SAFE_CALL(hipFree(subGpu));
    CUDA_SAFE_CALL(hipFree(rowGpu));
    CUDA_SAFE_CALL(hipFree(colGpu));
    CUDA_SAFE_CALL(hipFree(vBusGpu.mch));
    CUDA_SAFE_CALL(hipFree(vBusGpu.scr));
    CUDA_SAFE_CALL(hipFree(vBusGpu.aff));
    CUDA_SAFE_CALL(hipFree(hBusGpu));
    CUDA_SAFE_CALL(hipFree(resultsGpu));
    CUDA_SAFE_CALL(hipFree(bGpu));

    CUDA_SAFE_CALL(hipUnbindTexture(rowTexture));
    CUDA_SAFE_CALL(hipUnbindTexture(colTexture));
    CUDA_SAFE_CALL(hipUnbindTexture(hBusTexture));
    CUDA_SAFE_CALL(hipUnbindTexture(subTexture));

    free(params);

    //**************************************************************************

    TIMER_STOP;

    return NULL;
}

//------------------------------------------------------------------------------
//******************************************************************************

#endif // __HIPCC__
